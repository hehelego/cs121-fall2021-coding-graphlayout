#include "hip/hip_runtime.h"
#include "common.cuh"

#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <omp.h>

struct Vertex {
  Vec2D pos, disp;
  __host__ __device__ Vertex(const Vec2D &p) : pos(p), disp() {}
};

struct Edge {
  u32 u, v;
  __host__ __device__ Edge(u32 u = 0, u32 v = 0) : u(u), v(v) {}
  __host__ __device__ inline bool operator<(const Edge &e) const { return u == e.u ? v < e.v : u < e.u; }
};

// INPUT:
//   - vertices: array of vertex coordinates
//   - edges: array of edges in the graph
//   - edge_head,edge_tail: range(head[i], tail[i]) are the edges for i
//   - cr: repulsive force coefficient
//   - ca: attractive force coefficient
//   - N: number of vertices
//   - M: number of edges
//   - ITER: number of iterations to run
// OUTPUT:
//   - return: FP number, cost of time, in ms (1e-6 second)
FP layout(Vertex *vertices, Edge *edges, u32 *edge_head, u32 *edge_tail, FP cr, FP ca, u32 N, u32 M, u32 ITER) {
  Timer timer;
  timer.start();

  cr /= N, ca /= N;
  FP temperature = START_TEMPERATURE;

  for (u32 run = 0; run < ITER; run++) {
#pragma omp parallel for default(shared) schedule(dynamic, 16)
    for (u32 i = 0; i < N; i++) {
      // push away other vertices
      for (u32 j = 0; j < N; j++) {
        auto diff = vertices[j].pos - vertices[i].pos;
        auto dis2 = max2(MIN_DIS, diff.norm_square());
        auto dis1 = max2(MIN_DIS, sqrt(dis2));

        auto push = force_hookelastic(dis1, cr);
        vertices[j].disp += diff / dis1 * push;
      }
      // attract other vertices
      u32 l = edge_head[i], r = edge_tail[i];
      for (u32 k = l; k <= r; k++) {
        u32 j = edges[k].v;
        auto diff = vertices[j].pos - vertices[i].pos;
        auto dis2 = max2(MIN_DIS, diff.norm_square());
        auto dis1 = max2(MIN_DIS, sqrt(dis2));

        auto pull = force_gravitation(dis1, ca);
        vertices[j].disp -= diff / dis1 * pull;
      }
    }

    // move to new coordinates
#pragma omp parallel for default(shared) schedule(dynamic, 16)
    for (u32 i = 0; i < N; i++) {
      FP dis1 = sqrt(vertices[i].disp.norm_square());
      if (dis1 > temperature) vertices[i].pos += vertices[i].disp / dis1 * temperature;
      vertices[i].pos.x = min(Width / 2, max(-Width / 2, vertices[i].pos.x));
      vertices[i].pos.y = min(Height / 2, max(-Height / 2, vertices[i].pos.y));
      vertices[i].disp = Vec2D(0, 0);
    }

    temperature *= COOLING_FACTOR;
  }

  timer.end();
  return timer.delta();
}

i32 main(int argc, char *argv[]) {
  omp_set_num_threads(CPU_THS);

  if (argc < 3) {
    Debug() << "Usage: bin/cpu N M ITER in_file out_file\n";
    std::exit(1);
  }

  const u32 N = std::stoul(argv[1]), M = std::stoul(argv[2]), ITER = std::stoul(argv[3]);
  const String in_file(argv[4]), out_file(argv[5]);
  ifstream in_stream(in_file);
  ofstream out_stream(out_file);

  auto vertices = new Vertex[N];
  auto edges = new Edge[M];
  auto head = new u32[N + 1], tail = new u32[N + 1];
  std::fill(head, head + (N + 1), M);
  std::fill(tail, tail + (N + 1), 0);

  for (u32 i = 0, u, v; i < M; i++) {
    in_stream >> u >> v;
    if (u > v) swap2(u, v);
    edges[i] = Edge(u, v);
  }
  std::sort(edges, edges + M);
  for (u32 i = M - 1; i >= 0; i--) head[edges[i].u] = i;
  for (u32 i = 0; i < M; i++) tail[edges[i].u] = i;

  std::random_device rdev;
  std::mt19937 rng(rdev());
  std::uniform_real_distribution<f32> unif(-0.5, 0.5);
  for (u32 i = 0; i < N; i++) {
    FP x = unif(rng) * Width, y = unif(rng) * Height;
    vertices[i] = Vertex(x, y);
  }
  f32 K = sqrt((1.0 / N) * (Height * Width));
  layout(vertices, edges, head, tail, K, K, N, M, ITER);

  for (u32 i = 0; i < N; i++) {
    FP x = vertices[i].pos.x, y = vertices[i].pos.y;
    out_stream << x << ' ' << y << '\n';
  }
  delete[] vertices;
  delete[] edges;
  delete[] head, delete[] tail;
  return 0;
}
